#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32
#define BLOCK_DIM 16

__global__ void spmspm(COOMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {

    unsigned int r = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int temp = 0;


    if(r < A->numRows && c < B->numCols) {
        
        unsigned int rowPtrA = A->rowPtrs[r]; // Index of the current rowPtrs element
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;  // Number of non zero elements in A

        if(nnzA > 0){
            unsigned int *colIdxsA = A->colIdxs + rowPtrA;
            float *valueA = A->values + rowPtrA;

            // Loop over B columns
            unsigned int colPtrB = B->colPtrs[c];
            unsigned int nnzB = B->colPtrs[c + 1] - colPtrB;

            if(nnzB > 0){
                unsigned int *rowIdxsB = B->rowIdxs + colPtrB;
                float *valueB = B->values + colPtrB;

                // Loop and find intersection
                float sum = 0;
                unsigned int ia = 0;
                unsigned int ib = 0;

                // Loop over segment of non zero elements in the row of A and col of B
                while(ia < nnzA && ib < nnzB){
                    unsigned int colIdx = colIdxsA[ia];
                    unsigned int rowIdx = rowIdxsB[ib];
                    if(colIdx < rowIdx) {
                        ia++;
                    } else if(colIdx > rowIdx) {
                        ib++;
                    } else {
                        sum += valueA[ia] * valueB[ib];
                        ia++;
                        ib++;
                    }
                }
                // Sync threads
                // Write to Result
                if(sum > THRESHOLD || sum < -THRESHOLD) {
                    sum += bias;

                    //Remove negative and zero values
                    if(sum > 0) {
                        if(sum>YMAX) {
                            sum = YMAX;
                        }
                        
                        temp = atomicAdd(&result->nnz, 1);
                        result->colIdxs[temp] = c;
                        result->values[temp] = sum;
                        result->rowIdxs[temp] = r;
                    }
                }
            }
        }
    }                                    
}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

COOMatrix* createEmptyCOO_d(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    COOMatrix cooShadow;
    cooShadow.numRows = numRows;
    cooShadow.numCols = numCols;
    cooShadow.nnz = 0;
    cooShadow.capacity = capacity;
    hipMalloc((void**) &cooShadow.rowIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooShadow.values, capacity*sizeof(float));
    COOMatrix* coo_d;
    hipMalloc((void**) &coo_d, sizeof(COOMatrix));
    hipMemcpy(coo_d, &cooShadow, sizeof(COOMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return coo_d;
}

void copyCOOfromGPU(COOMatrix* coo_d, COOMatrix* coo) {
    COOMatrix cooShadow;
    hipMemcpy(&cooShadow, coo_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
    assert(coo->numRows == cooShadow.numRows);
    assert(coo->numCols == cooShadow.numCols);
    assert(coo->capacity >= cooShadow.nnz);
    coo->nnz = cooShadow.nnz;
    hipMemcpy(coo->rowIdxs, cooShadow.rowIdxs, cooShadow.nnz*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(coo->colIdxs, cooShadow.colIdxs, cooShadow.nnz*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(coo->values, cooShadow.values, cooShadow.nnz*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

CSRMatrix* createEmptyCSR_d(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    CSRMatrix csrShadow;
    csrShadow.numRows = numRows;
    csrShadow.numCols = numCols;
    csrShadow.nnz = 0;
    csrShadow.capacity = capacity;
    hipMalloc((void**) &csrShadow.rowPtrs, (numRows + 1)*sizeof(unsigned int));
    hipMalloc((void**) &csrShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &csrShadow.values, capacity*sizeof(float));
    CSRMatrix* csr_d;
    hipMalloc((void**) &csr_d, sizeof(CSRMatrix));
    hipMemcpy(csr_d, &csrShadow, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return csr_d;
}

void copyCSRtoGPU(CSRMatrix* csr, CSRMatrix* csr_d) {
    CSRMatrix csrShadow;
    hipMemcpy(&csrShadow, csr_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    assert(csrShadow.numRows == csr->numRows);
    assert(csrShadow.numCols == csr->numCols);
    assert(csrShadow.capacity >= csr->nnz);
    csrShadow.nnz = csr->nnz;
    hipMemcpy(csrShadow.rowPtrs, csr->rowPtrs, (csr->numRows + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrShadow.colIdxs, csr->colIdxs, csr->nnz*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrShadow.values, csr->values, csr->nnz*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

CSCMatrix* createCSCfromCSC_d(CSCMatrix* csc) {
    CSCMatrix cscShadow;
    cscShadow.numRows = csc->numRows;
    cscShadow.numCols = csc->numCols;
    cscShadow.nnz = csc->nnz;
    cscShadow.capacity = csc->capacity;
    hipMalloc((void**) &cscShadow.colPtrs, (csc->numCols + 1)*sizeof(unsigned int));
    hipMalloc((void**) &cscShadow.rowIdxs, csc->capacity*sizeof(unsigned int));
    hipMalloc((void**) &cscShadow.values, csc->capacity*sizeof(float));
    hipMemcpy(cscShadow.colPtrs, csc->colPtrs, (csc->numCols + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cscShadow.rowIdxs, csc->rowIdxs, csc->capacity*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cscShadow.values, csc->values, csc->capacity*sizeof(float), hipMemcpyHostToDevice);
    CSCMatrix* csc_d;
    hipMalloc((void**) &csc_d, sizeof(CSCMatrix));
    hipMemcpy(csc_d, &cscShadow, sizeof(CSCMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return csc_d;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createEmptyCSR(featureVectors->numRows, featureVectors->numCols, 4*featureVectors->nnz); // Assuming 4*nnz is enough for all Y vectors
    convertCOOtoCSR(featureVectors, Y0);
    CSRMatrix* Y0_d = createEmptyCSR_d(featureVectors->numRows, featureVectors->numCols, 4*featureVectors->nnz); // Assuming 4*nnz is enough for all Y vectors
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    CSCMatrix* W_d[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
        W_d[layer] = createCSCfromCSC_d(W[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Temporary buffer
    startTime(&timer);
    COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, Y0->capacity);
    COOMatrix *tmp_d = createEmptyCOO_d(Y0->numRows, Y0->numCols, Y0->capacity);
    stopTimeAndPrint(&timer, "Allocate temporary buffer");

    // Loop over layers
    CSRMatrix *Yin = Y0;
    COOMatrix *Yout = tmp;
    CSRMatrix *Yin_d = Y0_d;
    COOMatrix *Yout_d = tmp_d;
    // Configurations
    dim3 threadsPerBlock(BLOCK_DIM,BLOCK_DIM);
    dim3 blocksPerGrid((threadsPerBlock.x + Y0->numCols - 1)/threadsPerBlock.x,(threadsPerBlock.y + Y0->numRows - 1)/threadsPerBlock.y);

    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        printf("Computing layer %u (SpMSpM)\n", layer);
        
        // Copy to GPU
        startTime(&timer);
        copyCSRtoGPU(Yin, Yin_d);
        hipMemset(&Yout_d->nnz, 0, sizeof(unsigned int));
        stopTimeAndPrint(&timer, "    Copy CSR to GPU and clear COO");

        // SpMSpM
        startTime(&timer);
         spmspm <<< blocksPerGrid, threadsPerBlock >>> (Yout_d, Yin_d, W_d[layer], bias);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "    SpMSpM");

        // Copy from GPU
        startTime(&timer);
        copyCOOfromGPU(Yout_d, Yout);
        stopTimeAndPrint(&timer, "    Copy COO from GPU");
      

        // Convert COO to CSR
        startTime(&timer);
        convertCOOtoCSR(Yout, Yin);
        stopTimeAndPrint(&timer, "    Converting COO to CSR");
    

    }

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, Yin);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
    }
    freeCOO(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}

