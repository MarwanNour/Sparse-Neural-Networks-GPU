#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32

#define BLOCK_DIM 1024

__global__ spmspm(CSRMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {
    unsigned int nnzIdx = 0;


}


void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {


    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Double buffers
    startTime(&timer);
    CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, 2*Y0->nnz);
    CSRMatrix *inBuffer  = Y0;
    CSRMatrix *outBuffer = tmp;
    stopTimeAndPrint(&timer, "Allocate temporary buffer");
        
    // Allocate memory on GPU
    CSRMatrix *inBuffer_d;
    CSRMatrix *outBuffer_d;
    CSCMatrix *W_d;

    hipMalloc((void **) &inBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void **) &outBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void **) &W_d, sizeof(CSCMatrix));
    

    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        // Configurations
        const unsigned int threadsPerBlock = BLOCK_DIM;
        const unsigned int blocksPerGrid = (threadsPerBlock + outBuffer->numRows()*outBuffer->numCols() - 1)/threadsPerBlock;

        // Copy data to gpu
        hipMemcpy(inBuffer_d, inBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
        hipMemcpy(outBuffer_d, outBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
        hipMemcpy(W_d, W[layer], sizeof(CSRMatrix), hipMemcpyHostToDevice);

        // SpMSpM
        printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        // spmspm(outBuffer, inBuffer, W[layer], bias);
        spmspm <<< blocksPerGrid, threadsPerBlock >>>(outBuffer_d, inBuffer_d, W_d, bias);
        stopTimeAndPrint(&timer, "");

        // Swap buffers
        CSRMatrix *t = inBuffer;
        inBuffer = outBuffer;
        outBuffer = t;

        
    }
    // Free data on GPU
    hipFree(inBuffer_d);
    hipFree(outBuffer_d);
    hipFree(W_d);

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
    }
    freeCSR(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}

