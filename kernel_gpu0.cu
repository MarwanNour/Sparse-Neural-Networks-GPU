#include "hip/hip_runtime.h"

#include <stdio.h>

#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32

#define BLOCK_DIM 1024

__global__ void histogram_gpu(unsigned int* rowIdxs_input, unsigned int* rowPtrs_result, unsigned int numRows_input, unsigned int nnz_input){

    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    unsigned int size = numRows_input;

    // --------- Histogram ---------
    __shared__ unsigned int bins_s[10000];
    if(threadIdx.x < size){
        bins_s[threadIdx.x] = 0;
    }
    __syncthreads();

    while(i < nnz_input){
        unsigned char b = rowIdxs_input[i];
        atomicAdd(&bins_s[b], 1);
        i += stride;
    }
    __syncthreads();

    if(threadIdx.x < size){
        atomicAdd(&rowPtrs_result[threadIdx.x], bins_s[threadIdx.x]);
    }
}


__global__ void createCSRfromCOO_gpu(CSRMatrix* result, COOMatrix* A) {


    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Call histogram
    // histogram_gpu(A->rowIdxs, result->rowPtrs, A->numRows, A->nnz);

    // Prefix Sum
    // if(threadIdx.x == 0){
    //     unsigned int sum = 0;
    //     for(unsigned int row = 0; row < A->numRows; ++row) {
    //         unsigned int val = result->rowPtrs[row];
    //         result->rowPtrs[row] = sum;
    //         sum += val;
    //     }
    //     result->rowPtrs[A->numRows] = sum;
    // }
    thrust::exclusive_scan(thrust::device, result->rowPtrs, result->rowPtrs + result->numRows + 1, result->rowPtrs);
    __syncthreads();

    // Binning
    if(i == 0){
        for(unsigned int index = 0; index < A->nnz; ++index) {
            unsigned int row = A->rowIdxs[index];
            unsigned int i = result->rowPtrs[row]++;
            result->colIdxs[i] = A->colIdxs[index];
            result->values[i] = A->values[index];
        }


        // Restore row pointers
        for(unsigned int row = A->numRows - 1; row > 0; --row) {
            result->rowPtrs[row] = result->rowPtrs[row - 1];
        }

        result->rowPtrs[0] = 0;
        result->numRows = A->numRows;
        result->numCols = A->numCols;
        result->nnz = A->nnz;
        result->capacity = A->nnz;
    }

    if( i< A->numRows){
        int col_index =  result->rowPtrs[i];
        int col_index_one = result->rowPtrs[i + 1];

        thrust::sort_by_key(thrust::device, &(result->colIdxs[col_index]), &(result->colIdxs[col_index_one]), (result->values));
    }
    __syncthreads();

}

__global__ void spmspm(COOMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias, int *offset) {

    unsigned int r = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int nnzIdx = 0;
    unsigned int temp;
 
    if(r < A->numRows ){
        unsigned int rowPtrA = A->rowPtrs[r]; // Index of the current rowPtrs element
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;  // Number of non zero elements in A

        if(nnzA > 0){
            unsigned int *colIdxsA = A->colIdxs + rowPtrA;
            float *valueA = A->values + rowPtrA;

            // Loop over B columns
            for(unsigned int c = 0; c < B->numCols; ++c){
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] - colPtrB;

                if(nnzB > 0){
                    unsigned int *rowIdxsB = B->rowIdxs + colPtrB;
                    float *valueB = B->values + colPtrB;

                    // Loop and find intersection
                    float sum = 0;
                    unsigned int ia = 0;
                    unsigned int ib = 0;

                    // Loop over segment of non zero elements in the row of A and col of B
                    while(ia < nnzA && ib < nnzB){
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }
                    // Sync threads
                    // Write to Result
                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;

                        __syncthreads();

                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            nnzIdx++;
                            temp = atomicAdd(offset, 1);
                            result->colIdxs[temp] = c;
                            result->values[temp] = sum;
                            result->rowIdxs[temp] =r ;
                        }
                    }
                }
            }
        }
        atomicAdd(&result->nnz, nnzIdx);
    }
}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    CSRMatrix *inBuffer  = Y0;
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        CSCMatrix *W_h = createCSCfromCOO(layerWeights[layer]);
        CSCMatrix W_d;
        W_d.numRows = W_h->numRows;
        W_d.numCols = W_h->numCols;
        W_d.nnz = W_h->nnz;
        W_d.capacity = W_h->capacity;
        hipMalloc((void **) &W_d.colPtrs, (W_h->numCols + 1) * sizeof(unsigned int));
        hipMalloc((void **) &W_d.rowIdxs,W_h->nnz * sizeof(unsigned int));
        hipMalloc((void **) &W_d.values, W_h->nnz * sizeof(float));
        
        hipMalloc((void **) &W[layer], sizeof(CSCMatrix));
        // CSCMatrix *W_p_d;
        hipMemcpy(W_d.colPtrs, W_h->colPtrs, (W_h->numCols + 1)* sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(W_d.rowIdxs, W_h->rowIdxs, W_h->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(W_d.values, W_h->values, W_h->nnz * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(W[layer], &W_d, sizeof(CSCMatrix), hipMemcpyHostToDevice);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");


    

    // Allocate memory on GPU
    
    startTime(&timer);
    // ---------- in buffer ------------
    CSRMatrix inBuffer_d;
    inBuffer_d.numRows = inBuffer->numRows;
    inBuffer_d.numCols = inBuffer->numCols;
    inBuffer_d.nnz = inBuffer->nnz;
    inBuffer_d.capacity = inBuffer->capacity;
    hipMalloc((void **) &inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int));
    hipMalloc((void **) &inBuffer_d.colIdxs, inBuffer_d.capacity * sizeof(unsigned int));
    hipMalloc((void **) &inBuffer_d.values, inBuffer_d.capacity * sizeof(float));

    CSRMatrix *inBuffer_p_d;
    hipMalloc((void **) &inBuffer_p_d, sizeof(CSRMatrix));


    hipMemcpy(inBuffer_d.rowPtrs, inBuffer->rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(inBuffer_d.colIdxs, inBuffer->colIdxs, inBuffer->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(inBuffer_d.values, inBuffer->values, inBuffer->nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inBuffer_p_d, &inBuffer_d, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTimeAndPrint(&timer, "Allocate and copy inbuffer");

    startTime(&timer);
    // ----------- out buffer COO --------------
    //emptycoo
    COOMatrix* coo = (COOMatrix*) malloc(sizeof(COOMatrix));
    coo->numRows = inBuffer->numRows;
    coo->numCols = inBuffer->numCols;
    coo->nnz = inBuffer->nnz;
    coo->capacity = inBuffer_d.numRows * inBuffer_d.numCols;
    unsigned int *rowIdxs = (unsigned int *)malloc(inBuffer_d.numRows * inBuffer_d.numCols*sizeof(unsigned int));
    unsigned int *colIdxs = (unsigned int *)malloc(inBuffer_d.numRows * inBuffer_d.numCols*sizeof(unsigned int));
    float *values = (float *)malloc(inBuffer_d.numRows * inBuffer_d.numCols*sizeof(float));
    coo->rowIdxs = rowIdxs;
    coo->colIdxs = colIdxs;
    coo->values = values;

    COOMatrix outBufferCOO_d;
    outBufferCOO_d.numRows = inBuffer_d.numRows;
    outBufferCOO_d.numCols = inBuffer_d.numCols;
    outBufferCOO_d.nnz = inBuffer_d.nnz;
    outBufferCOO_d.capacity = inBuffer_d.numCols * inBuffer_d.numRows;
    hipMalloc((void **) &outBufferCOO_d.rowIdxs, inBuffer_d.numRows * inBuffer_d.numCols * sizeof(unsigned int));
    hipMalloc((void **) &outBufferCOO_d.colIdxs, inBuffer_d.numCols * inBuffer_d.numRows * sizeof(unsigned int));
    hipMalloc((void **) &outBufferCOO_d.values, inBuffer_d.numCols * inBuffer_d.numRows * sizeof(float));
    COOMatrix *outBufferCOO_p_d;
    hipMalloc((void **) &outBufferCOO_p_d, sizeof(COOMatrix));

    hipMemcpy(outBufferCOO_d.rowIdxs, coo->rowIdxs, coo->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(outBufferCOO_d.colIdxs, coo->colIdxs, coo->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(outBufferCOO_d.values, coo->values, coo->capacity * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(outBufferCOO_p_d, &outBufferCOO_d, sizeof(COOMatrix), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTimeAndPrint(&timer, "Allocate and copy coo");

    startTime(&timer);

    // ----------- out bufferCSR -----------
    CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, Y0->numRows*Y0->numCols);
    CSRMatrix outBufferCSR_d;
    outBufferCSR_d.numRows = inBuffer_d.numRows;
    outBufferCSR_d.numCols = inBuffer_d.numCols;
    // outBufferCSR_d.nnz = outBuffer->nnz;
    outBufferCSR_d.capacity = inBuffer_d.capacity;
    hipMalloc((void **) &outBufferCSR_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int));
    hipMalloc((void **) &outBufferCSR_d.colIdxs, inBuffer_d.capacity * sizeof(unsigned int));
    hipMalloc((void **) &outBufferCSR_d.values, inBuffer_d.capacity * sizeof(float));
    CSRMatrix *outBufferCSR_p_d;
    hipMalloc((void **) &outBufferCSR_p_d, sizeof(CSRMatrix));


    hipMemcpy(outBufferCSR_d.rowPtrs, tmp->rowPtrs, (tmp->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(outBufferCSR_d.colIdxs, tmp->colIdxs, tmp->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(outBufferCSR_d.values, tmp->values, tmp->capacity * sizeof(float), hipMemcpyHostToDevice);
  
    hipMemcpy(outBufferCSR_p_d, &outBufferCSR_d, sizeof(CSRMatrix), hipMemcpyHostToDevice);


    // -------------- W ----------------
    stopTimeAndPrint(&timer, "Allocate temporary csr");


    // Copy data from CPU to GPU
    hipDeviceSynchronize();
    

  


    // Configurations
    const unsigned int threadsPerBlock = BLOCK_DIM;
    const unsigned int blocksPerGrid = (threadsPerBlock + inBuffer->numRows - 1)/threadsPerBlock;
    int x=0;
    int *offset=&x ;
    
    CSRMatrix *t ;
    hipMalloc((void **) &t, sizeof(CSRMatrix));
    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        *offset= 0;
        // Copy W data to gpu
        startTime(&timer);
        // spmspm(outBuffer, inBuffer, W[layer], bias);
        // spmspm <<< blocksPerGrid, threadsPerBlock >>>(outBufferCOO_p_d, inBuffer_p_d, W[layer], bias,offset);
        

        stopTimeAndPrint(&timer, "layer");
        hipDeviceSynchronize();

        startTime(&timer);
      
        spmspm <<< blocksPerGrid, threadsPerBlock >>>(outBufferCOO_p_d, inBuffer_p_d, W[layer], bias,offset);
        hipDeviceSynchronize();

        stopTimeAndPrint(&timer, "spmspm");

        // printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        histogram_gpu<<< blocksPerGrid, threadsPerBlock >>>(outBufferCOO_p_d->rowIdxs, outBufferCSR_p_d->rowPtrs, outBufferCOO_p_d->numRows, outBufferCOO_p_d->nnz);
        hipDeviceSynchronize();

        createCSRfromCOO_gpu <<< blocksPerGrid, threadsPerBlock >>>(outBufferCSR_p_d, outBufferCOO_p_d);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "");

        // thrust::exclusive_scan(result->rowPtrs, result->rowPtrs + result->numRows, result->rowPtrs);

        // Swap buffers
       
        t = inBuffer_p_d;
        inBuffer_p_d = outBufferCSR_p_d;
        outBufferCSR_p_d = t;
    }

    // Copy data from GPU to CPU
    hipMemcpy(&inBuffer_d, inBuffer_p_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.nnz * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.nnz * sizeof(float), hipMemcpyDeviceToHost);

    // Free data on GPU
    // ---------- in buffer ------------
    hipFree(inBuffer_d.rowPtrs);
    hipFree(inBuffer_d.colIdxs);
    hipFree(inBuffer_d.values);
    hipFree(inBuffer_p_d);

    // ----------- out buffer COO -----------
    hipFree(outBufferCOO_d.rowIdxs);
    hipFree(outBufferCOO_d.colIdxs);
    hipFree(outBufferCOO_d.values);
    hipFree(outBufferCOO_p_d);

    // ----------- out bufferCSR -----------
    hipFree(outBufferCSR_d.rowPtrs);
    hipFree(outBufferCSR_d.colIdxs);
    hipFree(outBufferCSR_d.values);
    hipFree(outBufferCSR_p_d);

    // -------------- W ----------------
    // hipFree(W_d.colPtrs);
    // hipFree(W_d.rowIdxs);
    // hipFree(W_d.values);
    // hipFree(W_p_d);

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        hipFree(W[layer]);
        hipFree(W[layer]->colPtrs);
        hipFree(W[layer]->rowIdxs);
        hipFree(W[layer]->values);
    }
    // freeCSR(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}
 