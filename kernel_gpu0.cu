#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32

#define BLOCK_DIM 1024

__global__ void spmspm(CSRMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias, int offset) {

    unsigned int r = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int nnzIdx = 0;
    unsigned int temp;

    if(r < A->numRows ){
        unsigned int x=offset[r];
        unsigned int rowPtrA = A->rowPtrs[r]; // Index of the current rowPtrs element
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;  // Number of non zero elements in A

        if(nnzA > 0){
            unsigned int *colIdxsA = A->colIdxs + rowPtrA;
            float *valueA = A->values + rowPtrA;

            // Loop over B columns
            for(unsigned int c = 0; c < B->numCols; ++c){
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] = colPtrB;

                if(nnzB > 0){
                    unsigned int *rowIdxsB = B->rowIdxs + colPtrB;
                    float *valueB = B->values + colPtrB;

                    // Loop and find intersection
                    float sum = 0;
                    unsigned int ia = 0;
                    unsigned int ib = 0;

                    // Loop over segment of non zero elements in the row of A and col of B
                    while(ia < nnzA && ib < nnzB){
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }
                    // Sync threads
                    // Write to Result
                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;

                        __syncthreads();
                        
                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            nnzIdx++;
                            temp= atomicAdd(offset,1);
                            result->colIdxs[temp] = c;
                            result->values[temp] = sum;
                            result->rowIdxs[temp] =r ;
                        }    
                    }
                }
                // result->rowPtrs[r + 1] = x + temp; 
            }
        }
        // result->nnz = nnzIdx;  
        atomicAdd(&result->nnz, nnzIdx);     
    }

    // __syncthreads();


}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {


    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Double buffers
    startTime(&timer);
    CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, 2*Y0->nnz);
    CSRMatrix *inBuffer  = Y0;
    CSRMatrix *outBuffer = tmp;
    stopTimeAndPrint(&timer, "Allocate temporary buffer");
        
    // Allocate memory on GPU
    CSRMatrix *inBuffer_d;
    CSRMatrix *outBufferCOO_d;
    COOMatrix *outBuffer_d;
    CSCMatrix *W_d;

    hipMalloc((void **) &inBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void **) &outBufferCSR_d, sizeof(CSRMatrix));
    hipMalloc((void **) &W_d, sizeof(CSCMatrix));
    hipMalloc((void **) &outBufferCOO_d, sizeof(COOMatrix));

    hipMemcpy(inBuffer_d, inBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    // hipMemcpy(outBuffer_d, outBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        // Configurations
        const unsigned int threadsPerBlock = BLOCK_DIM;
        const unsigned int blocksPerGrid = (threadsPerBlock + outBuffer->numRows - 1)/threadsPerBlock;
        int offset=0;
        // Copy data to gpu
        hipMemcpy(W_d, W[layer], sizeof(CSRMatrix), hipMemcpyHostToDevice);

        // SpMSpM
        printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        // spmspm(outBuffer, inBuffer, W[layer], bias);
        spmspm <<< blocksPerGrid, threadsPerBlock >>>(outBufferCOO_d, inBuffer_d, W_d, bias,offset);
        stopTimeAndPrint(&timer, "");

        // printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        CooToCSR <<< blocksPerGrid, threadsPerBlock >>>(outBuffer_d, outBuffer1_d);
        // createCSRfromCOO(outBufferCSR_d,outBufferCOO_d)
        stopTimeAndPrint(&timer, "");

        // Swap buffers
        CSRMatrix *t = inBuffer_d;
        inBuffer_d = outBufferCSR_d;
        outBufferCSR_d = t;

        
    }
    // Free data on GPU
    hipMemcpy(inBuffer, inBuffer_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    // hipMemcpy(outBuffer, inBuffer_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);



    hipFree(inBuffer_d);
    hipFree(outBuffer_d);
    hipFree(W_d);

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
    }
    freeCSR(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}
