#include "hip/hip_runtime.h"

#include <stdio.h>

#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
} 
#define PRIVATE 4096
#define BLOCK_DIM 1024
// unsigned int* input, unsigned int* bins, unsigned int numElems
__global__ void histogram_gpu(CSRMatrix* result, COOMatrix* A){
    // {


        unsigned int* input ;
        unsigned int* bins;
        unsigned int numElems;
        input=A->rowIdxs;
        bins=result->rowPtrs;
        numElems=A->nnz;
        int tx = threadIdx.x; int bx = blockIdx.x;
        
        // compute global thread coordinates
        int i = (bx * blockDim.x) + tx;
        // if (i==0){printf(" histo nnz=%d",A->nnz);}

    // create a private histogram copy for each thread block
    __shared__ unsigned int hist[PRIVATE];
    if(bx==0){
        for (int j=tx; j<A->numRows; j+=BLOCK_DIM) {
                bins[j] = 0;
        }
    }

   

    // each thread must initialize more than 1 location
    if (PRIVATE > BLOCK_DIM) {
        for (int j=tx; j<PRIVATE; j+=BLOCK_DIM) {
            if (j < PRIVATE) {
                hist[j] = 0;
            }
        }
    }
    // use the first `PRIVATE` threads of each block to init
    else {
        if (tx < PRIVATE) {
            hist[tx] = 0;
        }
    }
    // wait for all threads in the block to finish
    __syncthreads();

    // update private histogram
    if (i < numElems) {
        if(input[i]<PRIVATE){
        atomicAdd(&(hist[input[i]]), 1);}
        else{
            atomicAdd(&(bins[input[i]]), 1);
        }
    }
    // wait for all threads in the block to finish
    __syncthreads();

    // each thread must update more than 1 location
    if (PRIVATE > BLOCK_DIM) {
        for (int j=tx; j<PRIVATE; j+=BLOCK_DIM) {
            if (j < PRIVATE) {
                atomicAdd(&(bins[j]), hist[j]);
            }
        }
    }
    // use the first `PRIVATE` threads to update final histogram
    else {
        if (tx < PRIVATE) {
            atomicAdd(&(bins[tx]), hist[tx]);
            
        }
    }
    // for(unsigned int i = 0; i < A->numRows+1; ++i) {
    //     // unsigned int row = A->rowIdxs[i];
    //     result->rowPtrs[i]=0;
    // }
    
    // for(unsigned int i = 0; i < A->nnz; ++i) {
    //             unsigned int row = A->rowIdxs[i];
    //             result->rowPtrs[row]++;
    //         }
}

__device__ void sort_by_key(unsigned int * keys,float * values,int size){
    int i, j,temp;
    float temp2;  
    for (i = 0; i < size-1; i++)   {   
      
    // Last i elements are already in place  
    for (j = 0; j < size-i-1; j++)  {
        if (keys[j] > keys[j+1]){
             temp = keys[j+ 1];  
            keys[j+1] = keys[j];  
            keys[j]= temp; 
            temp2 = keys[j+ 1];  
            keys[j+1] = keys[j];  
            keys[j]= temp2;  

            }
        }
    }
}
__global__ void Binning(CSRMatrix *result ,COOMatrix *A ){
    for(unsigned int index = 0; index < A->nnz; ++index) {
        unsigned int row = A->rowIdxs[index];
        unsigned int i = result->rowPtrs[row]++;
        result->colIdxs[i] = A->colIdxs[index];
        result->values[i] = A->values[index];
    }


    // Restore row pointers
    for(unsigned int row = A->numRows - 1; row > 0; --row) {
        result->rowPtrs[row] = result->rowPtrs[row - 1];
    }

    result->rowPtrs[0] = 0;
    result->numRows = A->numRows;
    result->numCols = A->numCols;
    result->nnz = A->nnz;
    result->capacity = A->capacity;
}

__global__ void createCSRfromCOO_gpu(CSRMatrix* result, COOMatrix* A) {


    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if(i==0){
    //     printf("nnz=%d\n",A->nnz);
    //     for(unsigned int i = 0; i < A->nnz; ++i) {
    //         unsigned int row = A->rowIdxs[i];
    //         result->rowPtrs[row]++;
    //     }


    // }
    // if(i==0){
    //     result->rowPtrs=0;
    // }
    // if(i<A->numRows){
    //     int count = thrust::count(thrust::device ,A->rowIdxs, A->rowIdxs+A->nnz, i);
    //     result->rowPtrs[i+1]=count;
    // }
    // __syncthreads();
// Call histogram
    // histogram_gpu(A->rowIdxs, result->rowPtrs, A->numRows, A->nnz);

    // Prefix Sum
    // if(threadIdx.x == 0){
    //     unsigned int sum = 0;
    //     for(unsigned int row = 0; row < A->numRows; ++row) {
    //         unsigned int val = result->rowPtrs[row];
    //         result->rowPtrs[row] = sum;
    //         sum += val;
    //     }
    //     result->rowPtrs[A->numRows] = sum;
    // }
   
    // __syncthreads();

    // Binning
    // if(i == 0){
       
    // }

    if( i< A->numRows){
        int col_index =  result->rowPtrs[i];
        int col_index_one = result->rowPtrs[i + 1];
        int size=col_index_one-col_index;

        sort_by_key( &(result->colIdxs[col_index]),&(result->values[col_index]),size);
    }
    // __syncthreads();
    if(i==0){
    A->nnz=0;
    }

}
__global__ void Prefix_sum(CSRMatrix *A){
    thrust::exclusive_scan(thrust::device, A->rowPtrs, A->rowPtrs + A->numRows + 1,A->rowPtrs );
}

__global__ void spmspm(COOMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {

    unsigned int r = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int nnzIdx = 0;
    unsigned int temp=0;

 if (r==0){
    printf("r= %d ",A->nnz);
 }
    if(r < A->numRows ){
        
        unsigned int rowPtrA = A->rowPtrs[r]; // Index of the current rowPtrs element
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;  // Number of non zero elements in A

        if(nnzA > 0){
            unsigned int *colIdxsA = A->colIdxs + rowPtrA;
            float *valueA = A->values + rowPtrA;

            // Loop over B columns
            for(unsigned int c = 0; c < B->numCols; ++c){
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] - colPtrB;

                if(nnzB > 0){
                    unsigned int *rowIdxsB = B->rowIdxs + colPtrB;
                    float *valueB = B->values + colPtrB;

                    // Loop and find intersection
                    float sum = 0;
                    unsigned int ia = 0;
                    unsigned int ib = 0;

                    // Loop over segment of non zero elements in the row of A and col of B
                    while(ia < nnzA && ib < nnzB){
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }
                    // Sync threads
                    // Write to Result
                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;

                        __syncthreads();

                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            nnzIdx++;
                            temp = atomicAdd(&result->nnz, 1);
                            result->colIdxs[temp] = c;
                            result->values[temp] = sum;
                            result->rowIdxs[temp] =r ;
                        }
                    }
                }
            }
        }
    }
}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    CSRMatrix *inBuffer  = Y0;
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        CSCMatrix *W_h = createCSCfromCOO(layerWeights[layer]);
        CSCMatrix W_d;
        W_d.numRows = W_h->numRows;
        W_d.numCols = W_h->numCols;
        W_d.nnz = W_h->nnz;
        W_d.capacity = W_h->capacity;
        hipMalloc((void **) &W_d.colPtrs, (W_h->numCols + 1) * sizeof(unsigned int));
        hipMalloc((void **) &W_d.rowIdxs,W_h->nnz * sizeof(unsigned int));
        hipMalloc((void **) &W_d.values, W_h->nnz * sizeof(float));
        
        hipMalloc((void **) &W[layer], sizeof(CSCMatrix));
        // CSCMatrix *W_p_d;
        hipMemcpy(W_d.colPtrs, W_h->colPtrs, (W_h->numCols + 1)* sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(W_d.rowIdxs, W_h->rowIdxs, W_h->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(W_d.values, W_h->values, W_h->nnz * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(W[layer], &W_d, sizeof(CSCMatrix), hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();
    stopTimeAndPrint(&timer, "Convert weights to CSR");


    

    // Allocate memory on GPU
    
    startTime(&timer);
    // ---------- in buffer ------------
    CSRMatrix inBuffer_d;
    inBuffer_d.numRows = inBuffer->numRows;
    inBuffer_d.numCols = inBuffer->numCols;
    inBuffer_d.nnz = inBuffer->nnz;
    inBuffer_d.capacity = inBuffer->numRows*inBuffer->numCols;
    hipMalloc((void **) &inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int));
    hipMalloc((void **) &inBuffer_d.colIdxs, inBuffer_d.capacity * sizeof(unsigned int));
    hipMalloc((void **) &inBuffer_d.values, inBuffer_d.capacity * sizeof(float));

    CSRMatrix *inBuffer_p_d;
    hipMalloc((void **) &inBuffer_p_d, sizeof(CSRMatrix));


    hipMemcpy(inBuffer_d.rowPtrs, inBuffer->rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(inBuffer_d.colIdxs, inBuffer->colIdxs, inBuffer->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(inBuffer_d.values, inBuffer->values, inBuffer->nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inBuffer_p_d, &inBuffer_d, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTimeAndPrint(&timer, "Allocate and copy inbuffer");

    startTime(&timer);
    // ----------- out buffer COO --------------
    //emptycoo
    COOMatrix* coo = (COOMatrix*) malloc(sizeof(COOMatrix));
    coo->numRows = inBuffer->numRows;
    coo->numCols = inBuffer->numCols;
    coo->nnz = inBuffer->nnz;
    coo->capacity = inBuffer_d.numRows * inBuffer_d.numCols;
    unsigned int *rowIdxs = (unsigned int *)malloc(inBuffer_d.numRows * inBuffer_d.numCols*sizeof(unsigned int));
    unsigned int *colIdxs = (unsigned int *)malloc(inBuffer_d.numRows * inBuffer_d.numCols*sizeof(unsigned int));
    float *values = (float *)malloc(inBuffer_d.numRows * inBuffer_d.numCols*sizeof(float));
    coo->rowIdxs = rowIdxs;
    coo->colIdxs = colIdxs;
    coo->values = values;

    COOMatrix outBufferCOO_d;
    outBufferCOO_d.numRows = inBuffer_d.numRows;
    outBufferCOO_d.numCols = inBuffer_d.numCols;
    outBufferCOO_d.nnz = 0;
    outBufferCOO_d.capacity = inBuffer_d.numCols * inBuffer_d.numRows;
    hipMalloc((void **) &outBufferCOO_d.rowIdxs, inBuffer_d.numRows * inBuffer_d.numCols * sizeof(unsigned int));
    hipMalloc((void **) &outBufferCOO_d.colIdxs, inBuffer_d.numCols * inBuffer_d.numRows * sizeof(unsigned int));
    hipMalloc((void **) &outBufferCOO_d.values, inBuffer_d.numCols * inBuffer_d.numRows * sizeof(float));
    COOMatrix *outBufferCOO_p_d;
    hipMalloc((void **) &outBufferCOO_p_d, sizeof(COOMatrix));

    gpuErrchk(hipMemcpy(outBufferCOO_d.rowIdxs, coo->rowIdxs, coo->capacity * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(outBufferCOO_d.colIdxs, coo->colIdxs, coo->capacity * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(outBufferCOO_d.values, coo->values, coo->capacity * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(outBufferCOO_p_d, &outBufferCOO_d, sizeof(COOMatrix), hipMemcpyHostToDevice));

    hipDeviceSynchronize();
    stopTimeAndPrint(&timer, "Allocate and copy coo");

    startTime(&timer);

    // ----------- out bufferCSR -----------
    // CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, Y0->numRows*Y0->numCols);
    // CSRMatrix outBufferCSR_d;
    // outBufferCSR_d.numRows = inBuffer_d.numRows;
    // outBufferCSR_d.numCols = inBuffer_d.numCols;
    // // outBufferCSR_d.nnz = outBuffer->nnz;
    // outBufferCSR_d.capacity = inBuffer_d.capacity;
    // hipMalloc((void **) &outBufferCSR_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int));
    // hipMalloc((void **) &outBufferCSR_d.colIdxs, inBuffer_d.capacity * sizeof(unsigned int));
    // hipMalloc((void **) &outBufferCSR_d.values, inBuffer_d.capacity * sizeof(float));
    // CSRMatrix *outBufferCSR_p_d;
    // hipMalloc((void **) &outBufferCSR_p_d, sizeof(CSRMatrix));


    // hipMemcpy(outBufferCSR_d.rowPtrs, tmp->rowPtrs, (tmp->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    // hipMemcpy(outBufferCSR_d.colIdxs, tmp->colIdxs, tmp->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    // hipMemcpy(outBufferCSR_d.values, tmp->values, tmp->capacity * sizeof(float), hipMemcpyHostToDevice);
  
    // gpuErrchk(hipMemcpy(outBufferCSR_p_d, &outBufferCSR_d, sizeof(CSRMatrix), hipMemcpyHostToDevice));


    // // -------------- W ----------------
    // stopTimeAndPrint(&timer, "Allocate temporary csr");


    // // Copy data from CPU to GPU
    // gpuErrchk(hipDeviceSynchronize());
    

  


    // Configurations
    const unsigned int threadsPerBlock = BLOCK_DIM;
    const unsigned int blocksPerGrid = (threadsPerBlock + inBuffer->numRows - 1)/threadsPerBlock;


    
    
    // CSRMatrix *t ;
    // hipMalloc((void **) &t, sizeof(CSRMatrix));
    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        printf("Computing layer %u (SpMSpM)\n", layer);


        // Copy W data to gpu
        // hipDeviceSynchronize();

        startTime(&timer);
      
        spmspm <<< blocksPerGrid, threadsPerBlock >>>(outBufferCOO_p_d, inBuffer_p_d, W[layer], bias);
        // hipDeviceSynchronize();
 
        gpuErrchk(hipDeviceSynchronize());

        stopTimeAndPrint(&timer, "spmspm");

       
        startTime(&timer);
        histogram_gpu<<< blocksPerGrid, threadsPerBlock  >>>( inBuffer_p_d,outBufferCOO_p_d);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "histogram done");

        startTime(&timer);
        Prefix_sum<<< 1, 1 >>>(inBuffer_p_d);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "prefix  done");

        startTime(&timer);
        Binning<<< 1, 1 >>>(inBuffer_p_d, outBufferCOO_p_d);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "binning  done");

        startTime(&timer);
        createCSRfromCOO_gpu <<< blocksPerGrid, threadsPerBlock >>>(inBuffer_p_d, outBufferCOO_p_d);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "csr convert done");

   

        // Swap buffers
     
       
        // t = inBuffer_p_d;
        // inBuffer_p_d = outBufferCSR_p_d;
        // outBufferCSR_p_d = t;
        // hipDeviceSynchronize();

    }

    // Copy data from GPU to CPU
    hipMemcpy(&inBuffer_d, inBuffer_p_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.nnz * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.nnz * sizeof(float), hipMemcpyDeviceToHost);

    // Free data on GPU
    // ---------- in buffer ------------
    hipFree(inBuffer_d.rowPtrs);
    hipFree(inBuffer_d.colIdxs);
    hipFree(inBuffer_d.values);
    hipFree(inBuffer_p_d);

    // ----------- out buffer COO -----------
    hipFree(outBufferCOO_d.rowIdxs);
    hipFree(outBufferCOO_d.colIdxs);
    hipFree(outBufferCOO_d.values);
    hipFree(outBufferCOO_p_d);

    // ----------- out bufferCSR -----------
    // hipFree(outBufferCSR_d.rowPtrs);
    // hipFree(outBufferCSR_d.colIdxs);
    // hipFree(outBufferCSR_d.values);
    // hipFree(outBufferCSR_p_d);

    // -------------- W ----------------
    // hipFree(W_d.colPtrs);
    // hipFree(W_d.rowIdxs);
    // hipFree(W_d.values);
    // hipFree(W_p_d);

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    // freeCSR(Y0);
    // for(unsigned int layer = 0; layer < numLayers; ++layer) {
    //     hipFree(W[layer]);
    //     hipFree(W[layer]->colPtrs);
    //     hipFree(W[layer]->rowIdxs);
    //     hipFree(W[layer]->values);
    // }
    // freeCSR(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}
 