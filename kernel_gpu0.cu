#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32

#define BLOCK_DIM 1024

__device__ void histogram(CSRMatrix* result, COOMatrix *A) {

    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    unsigned int size = A->numRows;

    // --------- Histogram ---------
    __shared__ unsigned int bins_s[size];
    if(threadIdx.x < size){
        bins_s[threadIdx.x] = 0;
    }
    __syncthreads();

    while(i < A->nnz){
        unsigned char b = A->rowIdxs[i];        
        atomicAdd(&bins_s[b], 1);
        i += stride;
    }
    __syncthreads();

    if(threadIdx.x < size){
        atomicAdd(&result->rowPtrs[threadIdx.x], bins_s[threadIdx.x]);
    }
}

__device__ void prefixSum(CSRMatrix* result, COOMatrix* A) {

    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    
    // --------- Kogge-Stone Exclusive -------


    /*
    // Prefix sum
    unsigned int sum = 0;
    for(unsigned int row = 0; row < A->numRows; ++row) {
        unsigned int val = rowPtrs[row];
        rowPtrs[row] = sum;
        sum += val;
    }
    rowPtrs[A->numRows] = sum;
    */
}

__device__ void createCSRfromCOO(CSRMatrix* result, COOMatrix* A) {
    histogram(result, A);

    prefixSum(result, A);

}

__global__ void spmspm(CSRMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {

    unsigned int r = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int nnzIdx0 = 0;
    unsigned int nnzIdx1 = 0; 
    __shared__ int offset[BLOCK_DIM];

    if(r < A->numRows ){
        unsigned int rowPtrA = A->rowPtrs[r]; // Index of the current rowPtrs element
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;  // Number of non zero elements in A

        if(nnzA > 0){
            unsigned int *colIdxsA = A->colIdxs + rowPtrA;
            float *valueA = A->values + rowPtrA;

            // Loop over B columns
            for(unsigned int c = 0; c < B->numCols; ++c){
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] = colPtrB;

                if(nnzB > 0){
                    unsigned int *rowIdxsB = B->rowIdxs + colPtrB;
                    float *valueB = B->values + colPtrB;

                    // Loop and find intersection
                    float sum = 0;
                    unsigned int ia = 0;
                    unsigned int ib = 0;

                    // Loop over segment of non zero elements in the row of A and col of B
                    while(ia < nnzA && ib < nnzB){
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }
                    // Sync threads
                    // Write to Result
                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;

                        // __syncthreads();
                        
                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            ++nnzIdx0;
                        }    
                    }
                }
            }
        }
        offset[r] = nnzIdx0;        
    }
    __syncthreads();
    
    // Prefix sum
    if(threadIdx.x == 0){
        if(r == 0){
            offset[r] = 0;
        }
        else{
            offset[0] = result->nnz;
        }
        for(int i = 1; i<BLOCK_DIM; ++i){
            offset[i] += offset[i-1];
        }
    }
    __syncthreads();
    

    if(r < A->numRows ){
        unsigned int x=offset[r];
        unsigned int rowPtrA = A->rowPtrs[r]; // Index of the current rowPtrs element
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;  // Number of non zero elements in A

        if(nnzA > 0){
            unsigned int *colIdxsA = A->colIdxs + rowPtrA;
            float *valueA = A->values + rowPtrA;

            // Loop over B columns
            for(unsigned int c = 0; c < B->numCols; ++c){
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] = colPtrB;

                if(nnzB > 0){
                    unsigned int *rowIdxsB = B->rowIdxs + colPtrB;
                    float *valueB = B->values + colPtrB;

                    // Loop and find intersection
                    float sum = 0;
                    unsigned int ia = 0;
                    unsigned int ib = 0;

                    // Loop over segment of non zero elements in the row of A and col of B
                    while(ia < nnzA && ib < nnzB){
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }
                    // Sync threads
                    // Write to Result
                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;

                        __syncthreads();
                        
                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            ++nnzIdx1;
                            result->colIdxs[nnzIdx1 + x] = c;
                            result->values[nnzIdx1 + x] = sum;
                        }    
                    }
                }
                result->rowPtrs[r + 1] = x + nnzIdx1; 
            }
        }
        // result->nnz = nnzIdx;  
        atomicAdd(&result->nnz, nnzIdx1);     
    }

    // __syncthreads();


}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {


    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Double buffers
    startTime(&timer);
    CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, 2*Y0->nnz);
    CSRMatrix *inBuffer  = Y0;
    CSRMatrix *outBuffer = tmp;
    stopTimeAndPrint(&timer, "Allocate temporary buffer");
        
    // Allocate memory on GPU
    CSRMatrix *inBuffer_d;
    CSRMatrix *outBuffer_d;
    CSCMatrix *W_d;

    hipMalloc((void **) &inBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void **) &outBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void **) &W_d, sizeof(CSCMatrix));
    

    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        // Configurations
        const unsigned int threadsPerBlock = BLOCK_DIM;
        const unsigned int blocksPerGrid = (threadsPerBlock + outBuffer->numRows - 1)/threadsPerBlock;

        // Copy data to gpu
        hipMemcpy(inBuffer_d, inBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
        hipMemcpy(outBuffer_d, outBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
        hipMemcpy(W_d, W[layer], sizeof(CSRMatrix), hipMemcpyHostToDevice);

        // SpMSpM
        printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        // spmspm(outBuffer, inBuffer, W[layer], bias);
        spmspm <<< blocksPerGrid, threadsPerBlock >>>(outBuffer_d, inBuffer_d, W_d, bias);
        stopTimeAndPrint(&timer, "");

        // Swap buffers
        CSRMatrix *t = inBuffer;
        inBuffer = outBuffer;
        outBuffer = t;

        
    }
    // Free data on GPU
    hipFree(inBuffer_d);
    hipFree(outBuffer_d);
    hipFree(W_d);

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
    }
    freeCSR(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}
